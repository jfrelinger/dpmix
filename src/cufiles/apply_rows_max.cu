
#include <hip/hip_runtime.h>

__global__ void
apply_rows_max(float* X, /** matrix to apply .. row major **/
	               	      float* y, /** result vector  **/
 			      int* iy, 
			      int rows,
			      int cols
  ) {

  unsigned int thidx = threadIdx.x;
  unsigned int thidy = threadIdx.y;
  unsigned int bid = blockIdx.x;
  unsigned int bdx = blockDim.x; // assumed equal to blockDim.y .. 16 or 32 ..

  int currow = bdx*bid;

  // flexible block size 
  extern __shared__ float shared_data[];
  float *sh_max = shared_data + bdx*bdx;

  if( thidy == 0 && thidx + currow < rows  ){
      sh_max[thidx] = -1e37;
  }
  __syncthreads();  
  
  float cur_val; float new_val; int argmax;
  for(int chunk = 0; chunk < cols; chunk+=bdx){
  	  // get some values chunking accross rows ...
	  if(currow+thidy < rows && chunk + thidx < cols){
	  	shared_data[thidx*bdx + thidy] = X[(currow + thidy)*cols + chunk + thidx];}
	  __syncthreads();
	  // get maximum in chunk ...
  	  if( thidy == 0 && thidx + currow < rows ){
	      for( int i = 0; i < bdx; i++){
	      	   if(chunk + i < cols){
	      	      cur_val = sh_max[thidx];
		      new_val = shared_data[i*bdx + thidx];
		      if( cur_val < new_val ){
		         sh_max[thidx] = new_val;
			 argmax = chunk + i;
		      }
                   }
	      }
	  }
	  __syncthreads();
  }
  // save values
  if(thidx + currow < rows && thidy==0){
    y[currow+thidx] = sh_max[thidx];
    iy[currow+thidx] = argmax;
  }

}

   